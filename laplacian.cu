#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include "book.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "eigenmap.h"
#include <matio.h>
#include <string>
using std::string;

__global__ void diag(double *dev_d, const double *dev_w, int n_patch);
__global__ void eye(double *dev_l, int n_patch);
__global__ void compute_l(double *dev_l, double *dev_w, int n_patch);
__device__ double atomicAdd(double* address, double val);
void diag_similarity_transform(hipblasHandle_t handle, double *dev_w, int n_patch);
void write_diag(const double *dev_d, int n_patch,
                string varname, string filename);
extern void write_laplacian(const double *dev_l, int n_patch,
                            string varname, string filename);
extern string filename;

/*
 * laplacian computes the Laplacian matrix based on the weight matrix.
 * dev_l: the laplacian matrix
 * dev_w: the weight matrix
 * n_patch: the dimension of dev_w and dev_l
 * Note: dev_l and dev_w are overwritten and stay (not freed) in the memory.
 */
void laplacian(double *dev_l, double *dev_w, int n_patch)
{
	/* ---- corresponding Matlab code ----
	 * D = diag(sum(W));
	 * L = eye(n_patch) - D^(-1/2)*W*D^(-1/2);
	 */
	//double *dev_d;
	hipblasHandle_t handle;
	//const double alpha = -1.0;
	//const double beta = 0.0;
	//const double one = 1.0;
	
	hipblasCreate(&handle);
	//HANDLE_ERROR(hipMalloc((void **)&dev_d, n_patch * n_patch * sizeof(double)));
	//HANDLE_ERROR(hipMemset(dev_d, 0, n_patch * n_patch * sizeof(double)));
	HANDLE_ERROR(hipMemset(dev_l, 0, n_patch * n_patch * sizeof(double)));
	//diag<<<BPG, TPB>>>(dev_d, dev_w, n_patch);

	eye<<<BPG, 1>>>(dev_l, n_patch); // L <- eye(n_patch)

#if 0 /* OLD CODE */
	// W <- (-1) * D * W + 0
    write_laplacian(dev_w, n_patch, "W_gpu_input",
                    filename + "_gpu_laplacian_input.mat");
	hipblasDsymm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, n_patch, n_patch, &alpha, dev_d, n_patch, dev_w, n_patch, &beta, dev_w, n_patch);

    write_laplacian(dev_w, n_patch, "W_gpu_intermediate",
                    filename + "_gpu_laplacian_intermediate.mat");
	// L <- 1 * W * D + L
	hipblasDsymm(handle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER, n_patch, n_patch, &one, dev_d, n_patch, dev_w, n_patch, &one, dev_l, n_patch);
#endif
    /* NEW */
    // W <- D * W * D
    write_laplacian(dev_w, n_patch, "W_gpu_input",
                    filename + "_gpu_laplacian_input.mat");
    diag_similarity_transform(handle, dev_w, n_patch);
    // L <- L - W
    compute_l<<<BPG, TPB>>>(dev_l, dev_w, n_patch);
    hipDeviceSynchronize();
	
#if 0
	//DEBUG
	double *l;
	int i, j;
	l = (double *)malloc(n_patch * n_patch * sizeof(double));
	HANDLE_ERROR(hipMemcpy(l, dev_l, n_patch * n_patch * sizeof(double), hipMemcpyDeviceToHost));
	for (i = 0; i < 15; i++) {
		for (j = 0; j < 15; j++)
			printf("%8.6f ", l[i + j * n_patch]);
		printf("\n");
	}
	free(l);
#endif
	//HANDLE_ERROR(hipFree(dev_d));
	hipblasDestroy(handle);
}

void diag_similarity_transform(hipblasHandle_t handle, double *dev_w, int n_patch)
{
    double *dev_d;
	HANDLE_ERROR(hipMalloc((void **)&dev_d, n_patch * sizeof(double)));
	HANDLE_ERROR(hipMemset(dev_d, 0, n_patch * sizeof(double)));

	diag<<<BPG, TPB>>>(dev_d, dev_w, n_patch);
    hipDeviceSynchronize();
    write_diag(dev_d, n_patch, "diag_gpu", filename + "_gpu_diag.mat");

    // row operations
    hipblasDdgmm(handle, HIPBLAS_SIDE_LEFT, n_patch, n_patch, dev_w, n_patch, dev_d, 1, dev_w, n_patch);
    hipDeviceSynchronize();
    // column operations
    hipblasDdgmm(handle, HIPBLAS_SIDE_RIGHT, n_patch, n_patch, dev_w, n_patch, dev_d, 1, dev_w, n_patch);
    hipDeviceSynchronize();

    HANDLE_ERROR(hipFree(dev_d));
}

__global__ void diag(double *dev_d, const double *dev_w, int n_patch)
{
	int b = blockIdx.x;
	int i, j;
	int size;
	__shared__ double cache[TPB];

	while (b < n_patch){
		size = TPB/2;
		i = threadIdx.x;
		j = i;
		cache[i] = 0.0;
		while (j < n_patch) {
			cache[i] += dev_w[b * n_patch + j];
			j += TPB;	
		}
		__syncthreads();
		while(size != 0){
			if (i < size){
				cache[i] += cache[i+size];
			}
			__syncthreads();
			size /= 2;
		}
		if (i == 0){
			dev_d[b] = 1/sqrt(cache[0]); // raise each diagonal entry to power of -1/2
		}
		__syncthreads();

		b += BPG;
	}
}

/*
__global__ void diag(double *dev_d, const double *dev_w, int n_patch)
{
	int b = blockIdx.x;
	int i, j;
	int size;
	__shared__ double cache[TPB];

	while (b < n_patch){
		size = TPB/2;
		i = threadIdx.x;
		j = i;
		cache[i] = 0.0;
		while (j < n_patch) {
			cache[i] += dev_w[b * n_patch + j];
			j += TPB;	
		}
		__syncthreads();
		while(size != 0){
			if (i < size){
				cache[i] += cache[i+size];
			}
			__syncthreads();
			size /= 2;
		}
		if (i == 0){
			dev_d[b + b*n_patch] = 1/sqrt(cache[0]); // raise each diagonal entry to power of -1/2
		}
		__syncthreads();

		b += BPG;
	}
}
*/
__global__ void eye(double *dev_l, int n_patch)
{
	int i = blockIdx.x;
	while(i < n_patch) {
		dev_l[i + i * n_patch] = 1.0;
		i += BPG;
	}
}
__global__ void compute_l(double *dev_l, double *dev_w, int n_patch)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int N = n_patch * n_patch;
    while (tid < N) {
        dev_l[tid] -= dev_w[tid];
        tid += blockDim.x * gridDim.x;
    }
}

__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}
void write_diag(const double *dev_d, int n_patch,
                string varname, string filename)
{
    mat_t *matfp;
    matvar_t *L;
    double *host_d = (double *)malloc(n_patch * sizeof(double));
	size_t d_dims[1] = {n_patch};

	matfp = Mat_CreateVer(filename.c_str(), NULL, MAT_FT_DEFAULT);

    hipMemcpy(host_d, dev_d, n_patch * sizeof(double), hipMemcpyDeviceToHost);
	L = Mat_VarCreate(varname.c_str(), MAT_C_DOUBLE, MAT_T_DOUBLE, 1, d_dims, host_d, 0);
    Mat_VarWrite(matfp, L, MAT_COMPRESSION_NONE);
    
	Mat_Close(matfp);
    Mat_VarFree(L);
    free(host_d);
}
